#include "hip/hip_runtime.h"
#include <chrono>
#include <random>
#include <iostream>
#include <cugemm_Mx2x2.hpp>

constexpr unsigned num_perf_test = 10;

namespace {
float fma(const float a, const float b, const float c) {return a * b + c;}
hipComplex fma(const hipComplex a, const hipComplex b, const hipComplex c) {return make_hipComplex(a.x * b.x - a.y * b.y + c.x, a.x * b.y + a.y * b.x + c.y);}

float mul(const float a, const float b) {return a * b;}
hipComplex mul(const hipComplex a, const hipComplex b) {return make_hipComplex(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);}

float sub(const float a, const float b) {return a - b;}
hipComplex sub(const hipComplex a, const hipComplex b) {return make_hipComplex(a.x - b.x, a.y - b.y);}

double norm2(const float a) {return a * a;};
double norm2(const hipComplex a) {return norm2(a.x) + norm2(a.y);};

bool is_zero(const float a) {return a == 0;}
bool is_zero(const hipComplex a) {return is_zero(a.x) && is_zero(a.y);}
template <class T>
T zero() {return 0;}
template <>
hipComplex zero<hipComplex>() {return make_hipComplex(0, 0);}

template <class T>
double gemm_Mx2x2_residual(
		const hipblasOperation_t op_a,
		const hipblasOperation_t op_b,
		const unsigned M,
		const T alpha,
		const T* const a_ptr, const std::size_t lda,
		const T* const b_ptr, const std::size_t ldb,
		const T beta,
		const T* const c_ptr, const std::size_t ldc,
		T* const t_ptr, const std::size_t ldt
		) {
	double diff_norm2 = 0.;
	double base_norm2 = 0.;
#pragma omp parallel for reduction(+: diff_norm2) reduction(+: base_norm2)
	for (unsigned m = 0; m < M; m++) {
		for (unsigned n = 0; n < 2; n++) {
			T c = zero<T>();
			for (unsigned k = 0; k < 2; k++) {
				const auto a_index = (op_a == HIPBLAS_OP_N) ? (m + lda * k) : (m * lda + k);
				const auto b_index = (op_b == HIPBLAS_OP_N) ? (k + ldb * n) : (k * ldb + n);

				c = fma(a_ptr[a_index], b_ptr[b_index], c);
			}
			if (is_zero(beta)) {
				c = mul(c, alpha);
			} else {
				c = fma(c, alpha, mul(c_ptr[m + n * ldc], beta));
			}
			base_norm2 += norm2(c);
			diff_norm2 += norm2(sub(c, t_ptr[m + n * ldt]));
		}
	}
	return std::sqrt(diff_norm2 / base_norm2);
}
} // unnamed namespace

enum gemm_mode_t {
	none,
	sgemm,
	cgemm,
};

int main(int argc, char** argv) {
	if (argc <= 4) {
		std::fprintf(stderr, "Usage: %s [GEMM mode (sgemm/cgemm)] [A Layout (N/T)] [B Layout (N/T)] [M] [(Optional) batch count]\n", argv[0]);
		return 1;
	}
	std::printf("mode,M,batch_size,residual,throughput_in_tflops,bw\n");

	const std::string gemm_mode_str = argv[1];
	auto gemm_mode = gemm_mode_t::none;
	if (gemm_mode_str == "sgemm") {
		gemm_mode = gemm_mode_t::sgemm;
	} else if (gemm_mode_str == "cgemm") {
		gemm_mode = gemm_mode_t::cgemm;
	}

	unsigned batch_size = 1;
	if (argc > 5) {
		batch_size = std::stoul(argv[5]);
	}


	const std::string op_a_str = argv[2];
	const std::string op_b_str = argv[3];
	hipblasOperation_t op_a, op_b;
	if (op_a_str == "N") {
		op_a = HIPBLAS_OP_N;
	} else if (op_a_str == "T") {
		op_a = HIPBLAS_OP_T;
	}
	if (op_b_str == "N") {
		op_b = HIPBLAS_OP_N;
	} else if (op_b_str == "T") {
		op_b = HIPBLAS_OP_T;
	}
	const auto M = std::stoul(argv[4]);

	auto mat_a_size = M * 2lu * batch_size * sizeof(float);
	auto mat_b_size = 2 * 2lu * batch_size * sizeof(float);
	auto mat_c_size = M * 2lu * batch_size * sizeof(float);

	std::size_t complexity = 2lu * 2 * 2 * M * batch_size;
	std::size_t num_elements = (2lu * 2lu + 2lu * M + 2lu * M) * batch_size;
	if (gemm_mode == gemm_mode_t::cgemm) {
		mat_a_size *= 2;
		mat_b_size *= 2;
		mat_c_size *= 2;
		num_elements *= 2;
		complexity *= 4lu;
	}

	float *host_mat_a, *host_mat_b, *host_mat_c, *host_mat_t;
	hipHostMalloc(&host_mat_a, mat_a_size);
	hipHostMalloc(&host_mat_b, mat_a_size);
	hipHostMalloc(&host_mat_c, mat_a_size);
	hipHostMalloc(&host_mat_t, mat_a_size);
	std::mt19937 mt(0);
	if (gemm_mode == gemm_mode_t::sgemm || gemm_mode_t::cgemm) {
		std::uniform_real_distribution<float> dist(-1, 1);
		for (std::size_t i = 0; i < mat_a_size / sizeof(float); i++) {
			host_mat_a[i] = dist(mt);
		}
		for (std::size_t i = 0; i < mat_b_size / sizeof(float); i++) {
			host_mat_b[i] = dist(mt);
		}
		for (std::size_t i = 0; i < mat_c_size / sizeof(float); i++) {
			host_mat_c[i] = 0;
		}
	}

	float *dev_mat_a, *dev_mat_b, *dev_mat_c;
	hipMalloc(&dev_mat_a, mat_a_size);
	hipMalloc(&dev_mat_b, mat_a_size);
	hipMalloc(&dev_mat_c, mat_a_size);
	hipMemcpy(dev_mat_a, host_mat_a, mat_a_size, hipMemcpyDefault);
	hipMemcpy(dev_mat_b, host_mat_b, mat_b_size, hipMemcpyDefault);
	hipMemcpy(dev_mat_c, host_mat_c, mat_c_size, hipMemcpyDefault);

	double elapsed_time_per_gemm = 0;
	double residual = 0;

	if (gemm_mode == gemm_mode_t::sgemm) {
		const auto alpha = 1.f;
		const auto beta = 0.f;
		if (batch_size == 1) {
			mtk::cugemm::gemm_Mx2x2(
					op_a, op_b,
					M,
					alpha,
					dev_mat_a, (op_a == HIPBLAS_OP_N ? M : 2),
					dev_mat_b, 2,
					beta,
					dev_mat_c, M
					);
		} else {
			mtk::cugemm::gemm_strided_batch_Mx2x2(
					op_a, op_b,
					M,
					alpha,
					dev_mat_a, (op_a == HIPBLAS_OP_N ? M : 2), M * 2,
					dev_mat_b, 2, 2 * 2,
					beta,
					dev_mat_c, M, M * 2,
					batch_size
					);
		}
		hipMemcpy(host_mat_t, dev_mat_c, mat_c_size, hipMemcpyDefault);
		for (unsigned b = 0; b < batch_size; b++) {
			residual += gemm_Mx2x2_residual(
					op_a, op_b,
					M,
					alpha,
					host_mat_a + M * 2 * b, (op_a == HIPBLAS_OP_N ? M : 2),
					host_mat_b + 2 * 2 * b, 2,
					beta,
					host_mat_c + M * 2 * b, M,
					host_mat_t + M * 2 * b, M
					);
		}
		residual /= batch_size;
		// throughput
		hipDeviceSynchronize();
		const auto start_clock = std::chrono::system_clock::now();
		for (unsigned i = 0; i < num_perf_test; i++) {
			if (batch_size == 1) {
				mtk::cugemm::gemm_Mx2x2(
						op_a, op_b,
						M,
						alpha,
						dev_mat_a, (op_a == HIPBLAS_OP_N ? M : 2),
						dev_mat_b, 2,
						beta,
						dev_mat_c, M
						);
			} else {
				mtk::cugemm::gemm_strided_batch_Mx2x2(
						op_a, op_b,
						M,
						alpha,
						dev_mat_a, (op_a == HIPBLAS_OP_N ? M : 2), M * 2,
						dev_mat_b, 2, 2 * 2,
						beta,
						dev_mat_c, M, M * 2,
						batch_size
						);
			}
		}
		hipDeviceSynchronize();
		const auto end_clock = std::chrono::system_clock::now();
		elapsed_time_per_gemm = std::chrono::duration_cast<std::chrono::nanoseconds>(end_clock - start_clock).count() * 1e-9 / num_perf_test;
	} else if (gemm_mode == gemm_mode_t::cgemm) {
		const auto alpha = make_hipComplex(1.f, 0.f);
		const auto beta = make_hipComplex(0.f, 0.f);
		if (batch_size == 1) {
			mtk::cugemm::gemm_Mx2x2(
					op_a, op_b,
					M,
					alpha,
					reinterpret_cast<hipComplex*>(dev_mat_a), (op_a == HIPBLAS_OP_N ? M : 2),
					reinterpret_cast<hipComplex*>(dev_mat_b), 2,
					beta,
					reinterpret_cast<hipComplex*>(dev_mat_c), M
					);
		} else {
			mtk::cugemm::gemm_strided_batch_Mx2x2(
					op_a, op_b,
					M,
					alpha,
					reinterpret_cast<hipComplex*>(dev_mat_a), (op_a == HIPBLAS_OP_N ? M : 2), M * 2,
					reinterpret_cast<hipComplex*>(dev_mat_b), 2, 2 * 2,
					beta,
					reinterpret_cast<hipComplex*>(dev_mat_c), M, M * 2,
					batch_size
					);
		}
		hipMemcpy(host_mat_t, dev_mat_c, mat_c_size, hipMemcpyDefault);
		for (unsigned b = 0; b < batch_size; b++) {
			residual += gemm_Mx2x2_residual(
					op_a, op_b,
					M,
					alpha,
					reinterpret_cast<hipComplex*>(host_mat_a) + M * 2 * b, (op_a == HIPBLAS_OP_N ? M : 2),
					reinterpret_cast<hipComplex*>(host_mat_b) + 2 * 2 * b, 2,
					beta,
					reinterpret_cast<hipComplex*>(host_mat_c) + M * 2 * b, M,
					reinterpret_cast<hipComplex*>(host_mat_t) + M * 2 * b, M
					);
		}
		residual /= batch_size;
		// throughput
		hipDeviceSynchronize();
		const auto start_clock = std::chrono::system_clock::now();
		for (unsigned i = 0; i < num_perf_test; i++) {
			if (batch_size == 1) {
				mtk::cugemm::gemm_Mx2x2(
						op_a, op_b,
						M,
						alpha,
						reinterpret_cast<hipComplex*>(dev_mat_a), (op_a == HIPBLAS_OP_N ? M : 2),
						reinterpret_cast<hipComplex*>(dev_mat_b), 2,
						beta,
						reinterpret_cast<hipComplex*>(dev_mat_c), M
						);
			} else {
				mtk::cugemm::gemm_strided_batch_Mx2x2(
						op_a, op_b,
						M,
						alpha,
						reinterpret_cast<hipComplex*>(dev_mat_a), (op_a == HIPBLAS_OP_N ? M : 2), M * 2,
						reinterpret_cast<hipComplex*>(dev_mat_b), 2, 2 * 2,
						beta,
						reinterpret_cast<hipComplex*>(dev_mat_c), M, M * 2,
						batch_size
						);
			}
		}
		hipDeviceSynchronize();
		const auto end_clock = std::chrono::system_clock::now();
		elapsed_time_per_gemm = std::chrono::duration_cast<std::chrono::nanoseconds>(end_clock - start_clock).count() * 1e-9 / num_perf_test;
	}

	const auto throughput_in_tflops = complexity / elapsed_time_per_gemm * 1e-12;
	const auto bandwidth_in_tb_per_s = num_elements * sizeof(float) / elapsed_time_per_gemm * 1e-12;

	std::printf("%s,%lu,%u,%e,%e,%e\n",
			gemm_mode_str.c_str(),
			M,
			batch_size,
			residual,
			throughput_in_tflops,
			bandwidth_in_tb_per_s
			);

	hipFree(dev_mat_a);
	hipFree(dev_mat_b);
	hipFree(dev_mat_c);

	hipHostFree(host_mat_a);
	hipHostFree(host_mat_b);
	hipHostFree(host_mat_c);
	hipHostFree(host_mat_t);
}
