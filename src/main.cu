#include "hip/hip_runtime.h"
#include <type_traits>
#include <algorithm>
#include <cassert>
#include <hip/hip_complex.h>
#include <cugemm_Mx2x2.hpp>

struct col_major;
struct row_major;

namespace detail {
bool is_zero(const float a) {return a == 0;}
bool is_zero(const hipComplex a) {return is_zero(a.x) && is_zero(a.y);}

__device__ float fma(const float a, const float b, const float c) {return fmaf(a, b, c);}
__device__ hipComplex fma(const hipComplex a, const hipComplex b, const hipComplex c) {
	return make_hipComplex(
			fma(a.x, b.x, fma(-a.y, b.y, c.x)),
			fma(a.y, b.x, fma( a.y, b.x, c.y))
			);
}

__device__ float mul(const float a, const float b) {return mul(a, b);}
__device__ hipComplex mul(const hipComplex a, const hipComplex b) {
	return make_hipComplex(
			fma(a.x, b.x, mul(-a.y, b.y)),
			fma(a.y, b.x, mul( a.y, b.x))
			);
}

template <class T>
__device__ T zero() {return 0.f;}
template <>
__device__ hipComplex zero<hipComplex>() {return make_hipComplex(0.f, 0.f);}
} // namespace detail

namespace {
template <class T, class LAYOUT_A, class LAYOUT_B, unsigned BLOCK_SIZE, unsigned M_PER_THREAD, bool BETA, unsigned N = 2, unsigned K = 2>
__device__ void gemm_core(
		const unsigned M,
		const T alpha,
		const T* const a_ptr, const std::size_t lda,
		const T* const b_ptr, const std::size_t ldb,
		const T beta,
		T* const c_ptr, const std::size_t ldc
		) {
	constexpr unsigned NUM_STAGES = 2;
	T frag_b[N * K];
	T frag_a[K * M_PER_THREAD * NUM_STAGES];
	T frag_c[N * M_PER_THREAD * NUM_STAGES];

	// Load B
	for (unsigned n = 0; n < N; n++) {
		for (unsigned k = 0; k < K; k++) {
			std::size_t index;
			if (std::is_same<LAYOUT_B, col_major>::value) {
				index = k + n * ldb;
			} else {
				index = n + k * ldb;
			}
			frag_b[k + n * K] = a_ptr[index];
		}
	}
	unsigned m_offset = M_PER_THREAD * (threadIdx.x + blockIdx.x * blockDim.x);
	if (m_offset >= M) {
		return;
	}

	// Load A
	for (unsigned m = 0; m < M_PER_THREAD; m++) {
		for (unsigned k = 0; k < K; k++) {
			std::size_t index;
			if (std::is_same<LAYOUT_A, col_major>::value) {
				index = m + k * lda;
			} else {
				index = k + m * lda;
			}
			frag_a[k + m * K] = a_ptr[index];
		}
	}

	// Load C
	if (!BETA) {
		for (unsigned m = 0; m < M_PER_THREAD; m++) {
			for (unsigned n = 0; n < N; n++) {
				std::size_t index = m + n * ldc;
				frag_c[n + m * N] = c_ptr[index];
			}
		}
	}

	//
	m_offset += M_PER_THREAD * gridDim.x * blockDim.x;
	unsigned stage = 1;
	for (; m_offset < M; m_offset += M_PER_THREAD * gridDim.x * blockDim.x) {
		// Load A
		for (unsigned m = 0; m < M_PER_THREAD; m++) {
			for (unsigned k = 0; k < K; k++) {
				std::size_t index;
				if (std::is_same<LAYOUT_A, col_major>::value) {
					index = (m + m_offset) + k * lda;
				} else {
					index = k + (m + m_offset) * lda;
				}
				frag_a[k + m * K + stage * M_PER_THREAD * K] = a_ptr[index];
			}
		}

		// Load C
		if (!BETA) {
			for (unsigned m = 0; m < M_PER_THREAD; m++) {
				for (unsigned n = 0; n < N; n++) {
					std::size_t index = m + m_offset + n * ldc;
					frag_c[n + m * N + stage * M_PER_THREAD * N] = c_ptr[index];
				}
			}
		}

		stage = 1 - stage;
		for (unsigned m = 0; m < M_PER_THREAD; m++) {
			for (unsigned n = 0; n < N; n++) {
				auto c = detail::zero<T>();
				for (unsigned k = 0; k < K; k++) {
					c = detail::fma(frag_a[m * K + k + stage * M_PER_THREAD * N], frag_b[n * K + k + stage * M_PER_THREAD * N], c);
				}
				if (BETA) {
					c = detail::fma(alpha, c, detail::mul(beta, frag_c[m + n * M_PER_THREAD + stage * M_PER_THREAD * N]));
				} else {
					c = detail::mul(alpha, c);
				}

				std::size_t index = m + m_offset + n * ldc - M_PER_THREAD * gridDim.x * blockDim.x;
				c_ptr[index] = c;
			}
		}
	}
	stage = 1 - stage;
	for (unsigned m = 0; m < M_PER_THREAD; m++) {
		for (unsigned n = 0; n < N; n++) {
			auto c = detail::zero<T>();
			for (unsigned k = 0; k < K; k++) {
				c = detail::fma(frag_a[m * K + k + stage * M_PER_THREAD * N], frag_b[n * K + k + stage * M_PER_THREAD * N], c);
			}
			if (BETA) {
				c = detail::fma(alpha, c, detail::mul(beta, frag_c[m + n * M_PER_THREAD + stage * M_PER_THREAD * N]));
			} else {
				c = detail::mul(alpha, c);
			}

			std::size_t index = m + m_offset + n * ldc - M_PER_THREAD * gridDim.x * blockDim.x;
			c_ptr[index] = c;
		}
	}
}

template <class T, class LAYOUT_A, class LAYOUT_B, unsigned BLOCK_SIZE, unsigned M_PER_THREAD, bool BETA, unsigned N = 2, unsigned K = 2>
__global__ void gemm_kernel(
		const unsigned M,
		const T alpha,
		const T* const a_ptr, const std::size_t lda,
		const T* const b_ptr, const std::size_t ldb,
		const T beta,
		T* const c_ptr, const std::size_t ldc
		) {
	gemm_core<T, LAYOUT_A, LAYOUT_B, BLOCK_SIZE, M_PER_THREAD, BETA, N, K>(M, alpha, a_ptr, lda, b_ptr, ldb, beta, c_ptr, ldc);
}

template <class T, class LAYOUT_A, class LAYOUT_B, unsigned BLOCK_SIZE, unsigned M_PER_THREAD, unsigned N = 2, unsigned K = 2>
void gemm_internal(
		const unsigned M,
		const T alpha,
		const T* const a_ptr, const std::size_t lda,
		const T* const b_ptr, const std::size_t ldb,
		const T beta,
		T* const c_ptr, const std::size_t ldc
		) {
	assert(M >= M_PER_THREAD);
	assert((M & (M - 1)) == 0);
	const auto min_grid_size = std::max<unsigned>(M / (M_PER_THREAD * BLOCK_SIZE), 1u);
	const auto grid_size = std::min<unsigned>(min_grid_size, 1024u);

	if (detail::is_zero(beta)) {
		gemm_kernel<T, LAYOUT_A, LAYOUT_B, BLOCK_SIZE, M_PER_THREAD, false, N, K><<<grid_size, BLOCK_SIZE>>>(M, alpha, a_ptr, lda, b_ptr, ldb, beta, c_ptr, ldc);
	} else {
		gemm_kernel<T, LAYOUT_A, LAYOUT_B, BLOCK_SIZE, M_PER_THREAD, true , N, K><<<grid_size, BLOCK_SIZE>>>(M, alpha, a_ptr, lda, b_ptr, ldb, beta, c_ptr, ldc);
	}
}
} // unnamed namespace

template <>
void mtk::cugemm::gemm_Mx2x2<float>(
		const hipblasOperation_t op_a,
		const hipblasOperation_t op_b,
		const unsigned M,
		const float alpha,
		const float* const a_ptr, const std::size_t lda,
		const float* const b_ptr, const std::size_t ldb,
		const float beta,
		float* const c_ptr, const std::size_t ldc
		) {
	if (op_a == HIPBLAS_OP_N && op_b == HIPBLAS_OP_N) {
		constexpr unsigned M_PER_THREAD = 4;
		constexpr unsigned BLOCK_SIZE = 256;
		gemm_internal<float, col_major, col_major, BLOCK_SIZE, M_PER_THREAD>(M, alpha, a_ptr, lda, b_ptr, ldb, beta, c_ptr, ldc);
	}
}

template <>
void mtk::cugemm::gemm_Mx2x2<hipComplex>(
		const hipblasOperation_t op_a,
		const hipblasOperation_t op_b,
		const unsigned M,
		const hipComplex alpha,
		const hipComplex* const a_ptr, const std::size_t lda,
		const hipComplex* const b_ptr, const std::size_t ldb,
		const hipComplex beta,
		hipComplex* const c_ptr, const std::size_t ldc
		) {
	if (op_a == HIPBLAS_OP_N && op_b == HIPBLAS_OP_N) {
		constexpr unsigned M_PER_THREAD = 4;
		constexpr unsigned BLOCK_SIZE = 256;
		gemm_internal<hipComplex, col_major, col_major, BLOCK_SIZE, M_PER_THREAD>(M, alpha, a_ptr, lda, b_ptr, ldb, beta, c_ptr, ldc);
	}
}
